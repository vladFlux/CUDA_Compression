#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <climits>
#include <iomanip>
#include <iostream>
#include <chrono>

#include "parallel_utilities.h"

/**
 * @file main_compress.cu
 * @brief Main entry point for GPU-accelerated Huffman compression system
 *
 * This file orchestrates the complete compression pipeline:
 * 1. File I/O and validation
 * 2. Character frequency analysis
 * 3. Huffman tree construction
 * 4. GPU resource analysis and optimization
 * 5. Compression execution
 * 6. Output file generation with embedded metadata
 *
 * The system automatically adapts to available GPU memory and file characteristics,
 * choosing optimal compression strategies without user intervention.
 */

// Minimum GPU scratch space required for safe operation (50MB)
// This ensures enough memory for temporary buffers and GPU operations
#define MIN_SCRATCH_SIZE (50 * 1024 * 1024)

/*=============================================================================
 * GLOBAL VARIABLE DEFINITIONS
 *=============================================================================*/

// Global instances of the core data structures defined in header
huffman_tree *head_huffman_tree_node; // Root of constructed Huffman tree
huffman_tree huffman_tree_node[512]; // Static array for all tree nodes
struct huffman_dictionary huffman_dictionary; // Main encoding lookup table
unsigned char bit_sequence_const_memory[256][255]; // Host storage for long bit sequences
unsigned int const_memory_flag = 0; // Flag for constant memory usage

/**
 * @brief Main compression program entry point
 * @param argc Number of command line arguments
 * @param argv Array of command line argument strings
 * @return EXIT_SUCCESS on successful compression, EXIT_FAILURE on error
 *
 * Implements the complete Huffman compression pipeline with automatic
 * GPU optimization and resource management. The program:
 *
 * 1. **File Processing**: Reads input file and validates arguments
 * 2. **Statistical Analysis**: Calculates character frequencies
 * 3. **Tree Construction**: Builds optimal Huffman encoding tree
 * 4. **GPU Analysis**: Determines optimal compression strategy based on:
 *    - Available GPU memory
 *    - File size and compression ratio
 *    - Integer overflow potential
 * 5. **Compression Execution**: Launches appropriate GPU kernels
 * 6. **Output Generation**: Creates compressed file with metadata
 *
 * The output file format includes:
 * - Original file length (4 bytes)
 * - Character frequency table (1024 bytes)
 * - Compressed data (variable length)
 *
 * This allows for complete decompression without external metadata.
 */
int main(const int argc, char **argv) {
    unsigned int index;
    unsigned int input_file_length, frequency[256];
    constexpr unsigned char bit_sequence_length = 0;
    unsigned char bit_sequence[255];
    long unsigned int mem_free, mem_total;

    /*=========================================================================
     * ARGUMENT VALIDATION AND FILE INPUT
     *=========================================================================*/

    // Validate command line arguments
    if (argc != 3) {
        std::cerr << "Invalid number of arguments." << std::endl <<
                "Example: <path_to_input_file> <path_to_output_file>" << std::endl;
        return EXIT_FAILURE;
    }

    // Read entire input file into memory
    // Using binary mode to handle all file types correctly
    FILE *input_file = fopen(argv[1], "rb");
    fseek(input_file, 0, SEEK_END); // Seek to end to get file size
    input_file_length = ftell(input_file); // Get file size in bytes
    fseek(input_file, 0, SEEK_SET); // Return to beginning for reading

    // Allocate memory buffer for entire file content
    auto *input_file_data = static_cast<unsigned char *>(malloc(input_file_length * sizeof(unsigned char)));
    fread(input_file_data, sizeof(unsigned char), input_file_length, input_file);
    fclose(input_file);

    /*=========================================================================
     * PERFORMANCE TIMING SETUP
     *=========================================================================*/

    // Start high-resolution timer for total execution time measurement
    const auto start = std::chrono::high_resolution_clock::now();

    /*=========================================================================
     * CHARACTER FREQUENCY ANALYSIS
     *=========================================================================*/

    // Initialize frequency array for all possible byte values (0-255)
    for (index = 0; index < 256; index++) {
        frequency[index] = 0;
    }

    // Count occurrence of each character in input data
    // This statistical analysis determines the optimal Huffman tree structure
    for (index = 0; index < input_file_length; index++) {
        frequency[input_file_data[index]]++;
    }

    /*=========================================================================
     * HUFFMAN TREE INITIALIZATION
     *=========================================================================*/

    // Create leaf nodes for each character that appears in the input
    // Only characters with non-zero frequency get nodes in the tree
    unsigned int distinct_character_count = 0;
    for (index = 0; index < 256; index++) {
        if (frequency[index] > 0) {
            huffman_tree_node[distinct_character_count].count = frequency[index];
            huffman_tree_node[distinct_character_count].letter = index;
            huffman_tree_node[distinct_character_count].left = nullptr; // Leaf nodes have no children
            huffman_tree_node[distinct_character_count].right = nullptr;
            distinct_character_count++;
        }
    }

    /*=========================================================================
     * HUFFMAN TREE CONSTRUCTION
     *=========================================================================*/

    // Build the binary tree by repeatedly combining lowest-frequency nodes
    // This implements the classic Huffman algorithm for optimal prefix codes
    for (index = 0; index < distinct_character_count - 1; index++) {
        const unsigned int combined_huffman_nodes = 2 * index;

        // Sort remaining nodes by frequency (lowest first)
        sort_huffman_tree(index, distinct_character_count, combined_huffman_nodes);

        // Combine the two lowest-frequency nodes into a new internal node
        build_huffman_tree(index, distinct_character_count, combined_huffman_nodes);
    }

    // Special case: if only one unique character exists, tree is just that character
    if (distinct_character_count == 1) {
        head_huffman_tree_node = &huffman_tree_node[0];
    }

    /*=========================================================================
     * HUFFMAN DICTIONARY GENERATION
     *=========================================================================*/

    // Traverse the completed tree to generate bit sequences for each character
    // Characters with higher frequency get shorter bit sequences
    build_huffman_dictionary(head_huffman_tree_node, bit_sequence, bit_sequence_length);

    /*=========================================================================
     * GPU MEMORY ANALYSIS AND OPTIMIZATION
     *=========================================================================*/

    // Query available GPU memory to determine compression strategy
    if (const hipError_t cuda_status = hipMemGetInfo(&mem_free, &mem_total); cuda_status != hipSuccess) {
        std::cerr << "Failed to get GPU memory info: " << hipGetErrorString(cuda_status) << std::endl;
        return EXIT_FAILURE;
    }

    // Display GPU memory information for user awareness
    std::cout << std::left << std::setw(25) << "Total GPU VRAM: " << std::right << std::setw(20) <<
            mem_total / (1024 * 1024) << " MB" << std::endl;
    std::cout << std::left << std::setw(25) << "Free GPU VRAM:  " << std::right << std::setw(20) <<
            mem_free / (1024 * 1024) << " MB" << std::endl;

    /*=========================================================================
     * COMPRESSION SIZE CALCULATION
     *=========================================================================*/

    // Calculate total compressed size in bits by summing each character's contribution
    // Each character contributes: frequency × bit_sequence_length
    long unsigned int mem_offset = 0;
    for (index = 0; index < 256; index++) {
        mem_offset += frequency[index] * huffman_dictionary.bit_sequence_length[index];
    }

    // Round up to nearest byte boundary for proper bit packing
    mem_offset = mem_offset % 8 == 0 ? mem_offset : mem_offset + 8 - mem_offset % 8;

    /*=========================================================================
     * GPU MEMORY REQUIREMENT CALCULATION
     *=========================================================================*/

    // Calculate fixed memory requirements for GPU compression:
    // - Input data array
    // - Bit offset array (input_file_length + 1 elements)
    // - Huffman dictionary structure
    const long unsigned int mem_data = input_file_length + (input_file_length + 1) * sizeof(unsigned int) + sizeof(
                                           huffman_dictionary);

    // Verify sufficient GPU memory exists for compression
    if (mem_free - mem_data < MIN_SCRATCH_SIZE) {
        printf("\nExiting : Not enough memory on GPU\nmem_free = %lu\nmin_mem_req = %lu\n", mem_free,
               mem_data + MIN_SCRATCH_SIZE);
        return EXIT_FAILURE;
    }

    /*=========================================================================
     * COMPRESSION STRATEGY DETERMINATION
     *=========================================================================*/

    // Calculate available memory for compressed data buffers (with 10MB safety margin)
    const long unsigned int mem_req = mem_free - mem_data - 10 * 1024 * 1024;

    // Determine number of kernel runs needed based on memory constraints
    // If compressed data fits in GPU memory: 1 run
    // If not: multiple runs with chunking
    const int num_kernel_runs = ceil(static_cast<double>(mem_offset) / mem_req);

    // Determine if integer overflow is possible in bit offset calculations
    // Check if memory requirements or compressed size could exceed UINT_MAX
    const unsigned int integer_overflow_flag = mem_req + 255 <= UINT_MAX || mem_offset + 255 <= UINT_MAX ? 0 : 1;

    /*=========================================================================
     * COMPRESSION STATISTICS DISPLAY
     *=========================================================================*/

    // Display compression information for user feedback
    std::cout << std::left << std::setw(25) << "Input file size: " << std::right << std::setw(20)
            << input_file_length << "  B" << std::endl;
    std::cout << std::left << std::setw(25) << "Compressed file size: " << std::right << std::setw(20)
            << mem_offset / 8 << "  B" << std::endl;

    /*=========================================================================
     * OFFSET ARRAY ALLOCATION AND COMPRESSION EXECUTION
     *=========================================================================*/

    // Allocate array for storing cumulative bit offsets
    // This array tells GPU threads exactly where to write compressed bits
    auto *compressed_data_offset = static_cast<unsigned int *>(malloc((input_file_length + 1) * sizeof(unsigned int)));

    // Launch the GPU compression pipeline
    // This function automatically handles all complexity:
    // - Offset array generation
    // - GPU memory management
    // - Kernel selection based on scenario
    // - Result retrieval
    launch_cuda_huffman_compress(input_file_data, compressed_data_offset, input_file_length, num_kernel_runs,
                                 integer_overflow_flag, mem_req);

    /*=========================================================================
     * PERFORMANCE MEASUREMENT
     *=========================================================================*/

    // Stop timer and calculate total execution time
    const auto end = std::chrono::high_resolution_clock::now();

    /*=========================================================================
     * COMPRESSED FILE OUTPUT
     *=========================================================================*/

    // Write compressed file with embedded metadata for decompression:
    // 1. Original file length (4 bytes) - needed to allocate decompression buffer
    // 2. Character frequency table (1024 bytes) - needed to reconstruct Huffman tree
    // 3. Compressed data (variable length) - the actual compressed content
    FILE *compressed_file = fopen(argv[2], "wb");
    fwrite(&input_file_length, sizeof(unsigned int), 1, compressed_file); // Original size
    fwrite(frequency, sizeof(unsigned int), 256, compressed_file); // Frequency table
    fwrite(input_file_data, sizeof(unsigned char), mem_offset / 8, compressed_file); // Compressed data
    fclose(compressed_file);

    /*=========================================================================
     * PERFORMANCE REPORTING
     *=========================================================================*/

    // Calculate and display execution time with millisecond precision
    const auto duration = std::chrono::duration<double>(end - start);
    const double total_seconds = duration.count();
    const int seconds = static_cast<int>(total_seconds);
    const int milliseconds = static_cast<int>((total_seconds - seconds) * 1000);

    std::cout << std::left << std::setw(25) << "Execution time: " << std::right << std::setw(15)
            << seconds << "s" << std::setw(5) << milliseconds << "ms" << std::endl;

    /*=========================================================================
     * CLEANUP AND EXIT
     *=========================================================================*/

    // Free all dynamically allocated memory
    free(input_file_data);
    free(compressed_data_offset);

    return EXIT_SUCCESS;
}
