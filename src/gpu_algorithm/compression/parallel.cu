#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include "parallel.h"


/**
 * @brief Sorts Huffman tree nodes by frequency using insertion sort
 * @param index_param Current iteration index in the tree building process
 * @param distinct_character_count Number of unique characters in input data
 * @param combined_huffman_nodes Starting index for nodes that haven't been combined yet
 *
 * This function implements insertion sort to arrange Huffman tree nodes in ascending order
 * by their frequency counts. The sorting is essential for the Huffman algorithm to work
 * correctly - we always want to combine the two nodes with the lowest frequencies.
 *
 * The sorting range is dynamic and shrinks with each iteration as nodes get combined
 * into the tree structure. Only uncombined nodes (from combined_huffman_nodes onward)
 * need to be sorted in each iteration.
 */
void sort_huffman_tree(const int index_param, const int distinct_character_count, const int combined_huffman_nodes) {
    // Define the range of nodes that need to be sorted
    const int start = combined_huffman_nodes;
    const int end = distinct_character_count - 1 + index_param;

    // Insertion sort: iterate through unsorted portion starting from second element
    for (int index = start + 1; index <= end; index++) {
        // Store the current element to be inserted into sorted portion
        const huffman_tree temp = huffman_tree_node[index];
        int sub_index = index - 1;

        // Shift elements in sorted portion that are greater than temp to the right
        // This creates space for inserting temp in its correct position
        while (sub_index >= start && huffman_tree_node[sub_index].count > temp.count) {
            huffman_tree_node[sub_index + 1] = huffman_tree_node[sub_index];
            sub_index--;
        }

        // Insert temp into its correct position in the sorted portion
        huffman_tree_node[sub_index + 1] = temp;
    }
}

/**
 * @brief Creates internal Huffman tree nodes by combining the two lowest-frequency nodes
 * @param index Current iteration in the tree building process
 * @param distinct_character_count Number of unique characters (leaf nodes)
 * @param combined_huffman_nodes Index of the first uncombined node
 *
 * This function implements the core of the Huffman algorithm by:
 * 1. Taking the two nodes with the lowest frequencies (after sorting)
 * 2. Creating a new internal node with their combined frequency
 * 3. Setting the new node's left and right children to point to these nodes
 * 4. Updating the tree head pointer to the newly created node
 *
 * The tree is built bottom-up, with leaf nodes representing individual characters
 * and internal nodes representing combined frequency groups. The final tree structure
 * determines the Huffman codes - more frequent characters get shorter paths from root.
 */
void build_huffman_tree(const int index, const int distinct_character_count, const int combined_huffman_nodes) {
    // Create new internal node by combining the two lowest-frequency nodes
    // The combined frequency is the sum of the two child frequencies
    huffman_tree_node[distinct_character_count + index].count =
            huffman_tree_node[combined_huffman_nodes].count + huffman_tree_node[combined_huffman_nodes + 1].count;

    // Set left child to point to the first (lowest frequency) node
    huffman_tree_node[distinct_character_count + index].left = &huffman_tree_node[combined_huffman_nodes];

    // Set right child to point to the second (second lowest frequency) node
    huffman_tree_node[distinct_character_count + index].right = &huffman_tree_node[combined_huffman_nodes + 1];

    // Update the tree head to point to this new internal node
    // The head always points to the most recently created internal node
    // After all iterations, head will point to the root of the complete tree
    head_huffman_tree_node = &(huffman_tree_node[distinct_character_count + index]);
}

/**
 * @brief Recursively traverses the Huffman tree to generate bit sequences for each character
 * @param root Current node being processed in the tree traversal
 * @param bit_sequence Array building the current bit sequence path
 * @param bit_sequence_length Current length of the bit sequence being built
 *
 * This function performs a depth-first traversal of the Huffman tree to generate
 * the binary codes for each character. The algorithm works as follows:
 *
 * - Left traversal adds a '0' bit to the current sequence
 * - Right traversal adds a '1' bit to the current sequence
 * - When a leaf node (character) is reached, the complete bit sequence is stored
 *
 * The function handles two memory storage strategies:
 * - Short sequences (<192 bits): stored in regular shared memory dictionary
 * - Long sequences (≥192 bits): split between shared memory and constant memory
 *
 * This dual storage approach optimizes GPU memory access patterns for different
 * sequence lengths, balancing memory bandwidth and cache efficiency.
 */
void build_huffman_dictionary(const huffman_tree *root, unsigned char *bit_sequence,
                              const unsigned char bit_sequence_length) {
    // Traverse left subtree (add '0' to bit sequence)
    if (root->left) {
        bit_sequence[bit_sequence_length] = 0;
        build_huffman_dictionary(root->left, bit_sequence, bit_sequence_length + 1);
    }

    // Traverse right subtree (add '1' to bit sequence)
    if (root->right) {
        bit_sequence[bit_sequence_length] = 1;
        build_huffman_dictionary(root->right, bit_sequence, bit_sequence_length + 1);
    }

    // Leaf node reached - store the complete bit sequence for this character
    if (root->left == nullptr && root->right == nullptr) {
        // Store the length of this character's bit sequence
        huffman_dictionary.bit_sequence_length[root->letter] = bit_sequence_length;

        if (bit_sequence_length < 192) {
            // Short sequence: store entirely in shared memory dictionary
            // This provides fastest access during GPU compression
            memcpy(huffman_dictionary.bit_sequence[root->letter], bit_sequence,
                   bit_sequence_length * sizeof(unsigned char));
        } else {
            // Long sequence: hybrid storage strategy
            // Store complete sequence in constant memory for full access
            memcpy(bit_sequence_const_memory[root->letter], bit_sequence, bit_sequence_length * sizeof(unsigned char));

            // Store first 191 bits in shared memory dictionary for fast initial access
            memcpy(huffman_dictionary.bit_sequence[root->letter], bit_sequence, 191);

            // Set global flag indicating constant memory is needed
            // This informs the GPU kernels to use hybrid memory access
            const_memory_flag = 1;
        }
    }
}

/**
 * @brief Generates bit offset array for simple single-kernel compression
 * @param compressed_data_offset Output array storing cumulative bit offsets
 * @param input_file_data Input data to analyze
 * @param input_file_length Size of input data in bytes
 *
 * This is the simplest offset calculation for optimal compression scenarios:
 * - Small to medium files that fit in GPU memory
 * - No integer overflow in cumulative bit calculations
 * - Single kernel launch will process entire file
 *
 * The offset array is crucial for parallel compression - it tells each GPU thread
 * exactly where to write the compressed bits for each input byte. Without
 * pre-calculated offsets, threads would need to synchronize constantly.
 *
 * The final offset is padded to byte boundary to ensure proper bit packing.
 */
void create_data_offset_array(unsigned int *compressed_data_offset, const unsigned char *input_file_data,
                              const unsigned int input_file_length) {
    // Initialize first offset to 0 (compression starts at bit 0)
    compressed_data_offset[0] = 0;

    // Calculate cumulative bit offsets for each input byte
    // Each byte's offset = previous offset + bit length of current byte's Huffman code
    for (int index = 0; index < input_file_length; index++) {
        compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]] +
                                            compressed_data_offset[index];
    }

    // Pad final offset to byte boundary if necessary
    // This ensures the compressed data aligns properly for bit packing
    // Example: if final bit offset is 13, pad to 16 (next multiple of 8)
    if (compressed_data_offset[input_file_length] % 8 != 0) {
        compressed_data_offset[input_file_length] = compressed_data_offset[input_file_length] + (
                                                        8 - (compressed_data_offset[input_file_length] % 8));
    }
}

/**
 * @brief Generates offset array with integer overflow detection and handling
 * @param compressed_data_offset Output bit offset array
 * @param input_file_data Input data to analyze
 * @param input_file_length Size of input data
 * @param integer_overflow_index Output array marking overflow positions
 * @param bit_padding_flag Output flags indicating bit padding needs at overflow points
 * @param num_bytes Safety margin to detect impending overflow
 *
 * This function handles compression scenarios where the cumulative bit offsets
 * exceed the range of unsigned integers (4.3 billion bits ≈ 537MB compressed).
 * This occurs with highly compressible data or very large files.
 *
 * When overflow is detected:
 * 1. The overflow position is recorded
 * 2. Bit alignment is checked and padding applied if needed
 * 3. Offset calculation restarts from 0 for post-overflow data
 *
 * The num_bytes parameter provides a safety margin (typically 8192) to detect
 * overflow before it occurs, preventing integer wraparound errors.
 */
void create_data_offset_array(unsigned int *compressed_data_offset, const unsigned char *input_file_data,
                              const unsigned int input_file_length, unsigned int *integer_overflow_index,
                              unsigned int *bit_padding_flag, const int num_bytes) {
    // Index for tracking multiple overflow points
    int sub_index = 0;
    compressed_data_offset[0] = 0;

    for (int index = 0; index < input_file_length; index++) {
        // Calculate next cumulative offset
        compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]] +
                                            compressed_data_offset[index];

        // Check for integer overflow (addition wraparound detection)
        // If next_offset + safety_margin < current_offset, overflow occurred
        if (compressed_data_offset[index + 1] + num_bytes < compressed_data_offset[index]) {
            // Record the position where overflow occurred
            integer_overflow_index[sub_index] = index;

            // Check if current position requires bit padding
            if (compressed_data_offset[index] % 8 != 0) {
                // Not on byte boundary - padding required
                bit_padding_flag[sub_index] = 1;

                // Calculate new offset with bit alignment consideration
                // Keep the remainder bits and add current byte's bit length
                compressed_data_offset[index + 1] =
                        (compressed_data_offset[index] % 8) + huffman_dictionary.bit_sequence_length
                        [input_file_data[index]];

                // Pad current offset to byte boundary
                compressed_data_offset[index] = compressed_data_offset[index] + (
                                                    8 - (compressed_data_offset[index] % 8));
            } else {
                // On byte boundary - no padding needed
                // Reset offset calculation starting from current byte's length
                compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]];
            }
            sub_index++;
        }
    }

    // Apply final byte boundary padding
    if (compressed_data_offset[input_file_length] % 8 != 0) {
        compressed_data_offset[input_file_length] = compressed_data_offset[input_file_length] + (
                                                        8 - (compressed_data_offset[input_file_length] % 8));
    }
}

/**
 * @brief Generates offset array for multi-kernel compression without integer overflow
 * @param compressed_data_offset Output bit offset array
 * @param input_file_data Input data to analyze
 * @param input_file_length Size of input data
 * @param gpu_memory_overflow_index Output array marking memory chunk boundaries
 * @param gpu_bit_padding_flag Output flags for bit padding at chunk boundaries
 * @param mem_req GPU memory limit for chunking decisions
 *
 * This function handles large files that must be split across multiple kernel
 * launches due to GPU memory constraints. The file is divided into chunks
 * that fit within available GPU memory.
 *
 * Key differences from single-run:
 * 1. Monitors memory usage instead of integer overflow
 * 2. Records chunk boundaries in gpu_memory_overflow_index
 * 3. Handles bit padding between chunks to maintain compression integrity
 * 4. Each chunk can be processed independently by separate kernel launches
 *
 * The chunking strategy ensures optimal GPU memory utilization while
 * maintaining compression efficiency across chunk boundaries.
 */
void create_data_offset_array(unsigned int *compressed_data_offset, const unsigned char *input_file_data,
                              const unsigned int input_file_length, unsigned int *gpu_memory_overflow_index,
                              unsigned int *gpu_bit_padding_flag, const long unsigned int mem_req) {
    int sub_index = 0;

    // Initialize chunk tracking arrays
    gpu_memory_overflow_index[0] = 0; // First chunk starts at index 0
    gpu_bit_padding_flag[0] = 0; // First chunk doesn't need padding
    compressed_data_offset[0] = 0;

    for (int index = 0; index < input_file_length; index++) {
        // Calculate cumulative bit offset
        compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]] +
                                            compressed_data_offset[index];

        // Check if current offset exceeds GPU memory limit
        if (compressed_data_offset[index + 1] > mem_req) {
            // Record chunk boundary: current chunk ends at position index
            gpu_memory_overflow_index[sub_index * 2 + 1] = index;
            // Next chunk starts at position index + 1
            gpu_memory_overflow_index[sub_index * 2 + 2] = index + 1;

            // Check bit alignment at chunk boundary
            if (compressed_data_offset[index] % 8 != 0) {
                // Chunk doesn't end on byte boundary - padding needed for next chunk
                gpu_bit_padding_flag[sub_index + 1] = 1;

                // Calculate offset for next chunk considering bit remainder
                compressed_data_offset[index + 1] =
                        (compressed_data_offset[index] % 8) + huffman_dictionary.bit_sequence_length
                        [input_file_data[index]];

                // Pad current chunk to byte boundary
                compressed_data_offset[index] = compressed_data_offset[index] + (
                                                    8 - (compressed_data_offset[index] % 8));
            } else {
                // Chunk ends on byte boundary - clean break
                compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]];
            }
            sub_index++;
        }
    }

    // Apply final padding and record final chunk boundary
    if (compressed_data_offset[input_file_length] % 8 != 0) {
        compressed_data_offset[input_file_length] = compressed_data_offset[input_file_length] + (
                                                        8 - (compressed_data_offset[input_file_length] % 8));
    }
    gpu_memory_overflow_index[sub_index * 2 + 1] = input_file_length;
}

/**
 * @brief Generates offset array for the most complex scenario: multi-kernel with integer overflow
 * @param compressed_data_offset Output bit offset array
 * @param input_file_data Input data to analyze
 * @param input_file_length Size of input data
 * @param integer_overflow_index Output array for integer overflow positions
 * @param bit_padding_flag Output flags for overflow padding
 * @param gpu_memory_overflow_index Output array for memory chunk boundaries
 * @param gpu_bit_padding_flag Output flags for chunk padding
 * @param num_bytes Safety margin for overflow detection
 * @param mem_req GPU memory limit
 *
 * This is the most complex offset calculation, handling both:
 * 1. Memory-based chunking for large files
 * 2. Integer overflow within chunks for highly compressible data
 *
 * The function must coordinate two different types of boundaries:
 * - Memory boundaries: where chunks are split due to GPU memory limits
 * - Overflow boundaries: where integer arithmetic overflows within chunks
 *
 * Special logic handles the interaction between these two boundary types,
 * ensuring that both memory management and overflow recovery work correctly
 * when they occur in the same compression job.
 *
 * This scenario typically occurs with very large, highly compressible files
 * that require both multi-kernel processing and overflow handling.
 */
void create_data_offset_array(unsigned int *compressed_data_offset, const unsigned char *input_file_data,
                              const unsigned int input_file_length, unsigned int *integer_overflow_index,
                              unsigned int *bit_padding_flag, unsigned int *gpu_memory_overflow_index,
                              unsigned int *gpu_bit_padding_flag, const int num_bytes,
                              const long unsigned int mem_req) {
    int sub_index = 0; // Counter for integer overflow events
    int overflow_index = 0; // Counter for memory overflow (chunk) events
    compressed_data_offset[0] = 0;

    for (int index = 0; index < input_file_length; index++) {
        // Calculate next cumulative offset
        compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]] +
                                            compressed_data_offset[index];

        // Check for memory limit exceeded (but only if we've had integer overflow before)
        // This complex condition handles the interaction between memory chunking and integer overflow
        if (sub_index != 0 && (static_cast<long unsigned int>(compressed_data_offset[index + 1]) + compressed_data_offset[
                           integer_overflow_index[sub_index - 1]] > mem_req)) {
            // Memory limit exceeded - create new chunk boundary
            gpu_memory_overflow_index[overflow_index * 2 + 1] = index;
            gpu_memory_overflow_index[overflow_index * 2 + 2] = index + 1;

            // Handle bit padding for chunk boundary
            if (compressed_data_offset[index] % 8 != 0) {
                gpu_bit_padding_flag[overflow_index + 1] = 1;
                compressed_data_offset[index + 1] =
                        (compressed_data_offset[index] % 8) + huffman_dictionary.bit_sequence_length
                        [input_file_data[index]];
                compressed_data_offset[index] = compressed_data_offset[index] + (
                                                    8 - (compressed_data_offset[index] % 8));
            } else {
                compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]];
            }
            overflow_index++;
        } else if (compressed_data_offset[index + 1] + num_bytes < compressed_data_offset[index]) {
            // Integer overflow detected - handle overflow boundary
            integer_overflow_index[sub_index] = index;

            // Handle bit padding for overflow boundary
            if (compressed_data_offset[index] % 8 != 0) {
                bit_padding_flag[sub_index] = 1;
                compressed_data_offset[index + 1] =
                        (compressed_data_offset[index] % 8) + huffman_dictionary.bit_sequence_length
                        [input_file_data[index]];
                compressed_data_offset[index] = compressed_data_offset[index] + (
                                                    8 - (compressed_data_offset[index] % 8));
            } else {
                compressed_data_offset[index + 1] = huffman_dictionary.bit_sequence_length[input_file_data[index]];
            }
            sub_index++;
        }
    }

    // Apply final padding and record final boundaries
    if (compressed_data_offset[input_file_length] % 8 != 0) {
        compressed_data_offset[input_file_length] = compressed_data_offset[input_file_length] + (
                                                        8 - (compressed_data_offset[input_file_length] % 8));
    }
    gpu_memory_overflow_index[sub_index * 2 + 1] = input_file_length;
}
