#include "hip/hip_runtime.h"
#include "parallel_utilities.h"


/**
 * @brief CUDA kernel for single-run compression without integer overflow
 * @param d_input_file_data Device array containing the raw input data to compress
 * @param d_compressed_data_offset Device array with pre-calculated bit offsets for each byte
 * @param d_huffman_dictionary Device copy of the Huffman encoding table
 * @param d_byte_compressed_data Device buffer for intermediate bit-level compressed data
 * @param d_input_file_length Length of input data in bytes
 * @param const_memory_flag Flag indicating whether to use constant memory for long bit sequences
 *
 * This kernel handles the optimal compression case:
 * - Small to medium files that fit entirely in GPU memory
 * - No integer overflow in bit offset calculations
 * - Single kernel launch processes entire file
 *
 * The compression process occurs in two phases:
 * 1. Bit-level encoding: Each input byte is replaced with its Huffman bit sequence
 * 2. Bit packing: Groups of 8 bits are packed into output bytes
 */
__global__ void compress(unsigned char *d_input_file_data, const unsigned int *d_compressed_data_offset,
                         const struct huffman_dictionary *d_huffman_dictionary, unsigned char *d_byte_compressed_data,
                         const unsigned int d_input_file_length, const unsigned int const_memory_flag) {
    // Copy Huffman dictionary to shared memory for fast access across all threads in block
    // Shared memory provides much faster access than global memory for frequently used data
    __shared__ struct huffman_dictionary table;
    memcpy(&table, d_huffman_dictionary, sizeof(struct huffman_dictionary));

    const unsigned int input_file_length = d_input_file_length;
    unsigned int index, bit_index;

    // Calculate unique thread ID within the grid
    const unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Phase 1: Convert each input byte to its Huffman bit sequence
    // Two paths based on whether constant memory is needed for very long bit sequences
    if (const_memory_flag == 0) {
        // Standard path: All bit sequences fit in shared memory
        // Each thread processes every (blockDim.x)th element to ensure coalesced memory access
        for (index = pos; index < input_file_length; index += blockDim.x) {
            // For each input byte, copy its Huffman bit sequence to the compressed data buffer
            // d_compressed_data_offset[index] gives the bit position where this byte's encoding starts
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                    d_input_file_data[index]][bit_index];
            }
        }
    } else {
        // Hybrid path: Use both shared memory and constant memory
        // For very long bit sequences (>191 bits), use constant memory for the overflow
        for (index = pos; index < input_file_length; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                if (bit_index < 191) {
                    // Short sequences: use fast shared memory
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                        d_input_file_data[index]][
                        bit_index];
                } else {
                    // Long sequences: use constant memory for bits beyond 191
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = d_bit_sequence_const_memory[
                        d_input_file_data[index]][bit_index];
                }
            }
        }
    }

    // Synchronize all threads before proceeding to bit packing phase
    // Ensures all bit sequences are written before packing begins
    __syncthreads();

    // Phase 2: Pack individual bits into bytes
    // Each thread processes 8 bits (1 byte) at a time
    // pos * 8 ensures each thread starts at a different 8-bit boundary
    for (index = pos * 8; index < d_compressed_data_offset[input_file_length]; index += blockDim.x * 8) {
        // Process 8 consecutive bits and pack them into a single output byte
        for (unsigned int sub_index = 0; sub_index < 8; sub_index++) {
            if (d_byte_compressed_data[index + sub_index] == 0) {
                // Bit is 0: shift left and add 0 (just shift)
                d_input_file_data[index / 8] = d_input_file_data[index / 8] << 1;
            } else {
                // Bit is 1: shift left and set LSB to 1
                d_input_file_data[index / 8] = (d_input_file_data[index / 8] << 1) | 1;
            }
        }
    }
}

/**
 * @brief CUDA kernel for single-run compression with integer overflow handling
 * @param d_input_file_data Device input data array (reused for output)
 * @param d_compressed_data_offset Device array with bit offsets
 * @param d_huffman_dictionary Device Huffman encoding table
 * @param d_byte_compressed_data Device buffer for pre-overflow compressed bits
 * @param d_temp_overflow Device buffer for post-overflow compressed bits
 * @param d_input_file_length Length of input data
 * @param const_memory_flag Flag for constant memory usage
 * @param overflow_position Index where integer overflow occurs in offset array
 *
 * This kernel handles compression when bit offsets exceed unsigned int range.
 * The compression is split at the overflow point:
 * - Data before overflow goes to d_byte_compressed_data
 * - Data after overflow goes to d_temp_overflow
 * - Both segments are then packed separately and concatenated
 *
 * This scenario occurs with highly compressible data where the cumulative
 * bit offsets grow beyond what can be represented in 32-bit integers.
 */
__global__ void compress(unsigned char *d_input_file_data, const unsigned int *d_compressed_data_offset,
                         const struct huffman_dictionary *d_huffman_dictionary, unsigned char *d_byte_compressed_data,
                         unsigned char *d_temp_overflow, const unsigned int d_input_file_length,
                         const unsigned int const_memory_flag, const unsigned int overflow_position) {
    // Copy Huffman table to shared memory for fast access
    __shared__ struct huffman_dictionary table;
    memcpy(&table, d_huffman_dictionary, sizeof(struct huffman_dictionary));

    const unsigned int input_file_length = d_input_file_length;
    unsigned int index, sub_index, bit_index;
    const unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Phase 1: Bit-level encoding with overflow handling
    if (const_memory_flag == 0) {
        // Process data before overflow point
        // This data uses normal offset calculations
        for (index = pos; index < overflow_position; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                    d_input_file_data[index]][bit_index];
            }
        }

        // Process data after overflow point
        // Skip the overflow byte itself (handled separately) and process remaining data
        for (index = overflow_position + pos; index < input_file_length - 1; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index + 1]]; bit_index++) {
                d_temp_overflow[d_compressed_data_offset[index + 1] + bit_index] = table.bit_sequence[d_input_file_data[
                    index + 1]][bit_index];
            }
        }

        // Handle the overflow byte specially (only thread 0 does this to avoid race conditions)
        // Places the overflow byte's bit sequence at the boundary between buffers
        if (pos == 0) {
            memcpy(
                &d_temp_overflow[d_compressed_data_offset[(overflow_position + 1)] - table.bit_sequence_length[
                                     d_input_file_data[overflow_position]]],
                &table.bit_sequence[d_input_file_data[overflow_position]],
                table.bit_sequence_length[d_input_file_data[overflow_position]]);
        }
    } else {
        // Hybrid memory approach with overflow handling
        // Process pre-overflow data using shared/constant memory
        for (index = pos; index < overflow_position; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                if (bit_index < 191) {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                        d_input_file_data[index]][
                        bit_index];
                } else {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = d_bit_sequence_const_memory[
                        d_input_file_data[index]][bit_index];
                }
            }
        }

        // Process post-overflow data using shared/constant memory
        for (index = overflow_position + pos; index < input_file_length - 1; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index + 1]]; bit_index++) {
                if (bit_index < 191) {
                    d_temp_overflow[d_compressed_data_offset[index + 1] + bit_index] = table.bit_sequence[
                        d_input_file_data[index + 1]][
                        bit_index];
                } else {
                    d_temp_overflow[d_compressed_data_offset[index + 1] + bit_index] = d_bit_sequence_const_memory[
                        d_input_file_data
                        [index + 1]][bit_index];
                }
            }
        }

        // Handle overflow byte using constant memory if sequence is long
        if (pos == 0) {
            memcpy(
                &d_temp_overflow[d_compressed_data_offset[(overflow_position + 1)] - table.bit_sequence_length[
                                     d_input_file_data[overflow_position]]],
                &d_bit_sequence_const_memory[d_input_file_data[overflow_position]],
                table.bit_sequence_length[d_input_file_data[overflow_position]]);
        }
    }

    // Ensure all bit sequences are written before packing
    __syncthreads();

    // Phase 2: Bit packing for pre-overflow data
    // Pack bits from d_byte_compressed_data into the beginning of output buffer
    for (index = pos * 8; index < d_compressed_data_offset[overflow_position]; index += blockDim.x * 8) {
        for (sub_index = 0; sub_index < 8; sub_index++) {
            if (d_byte_compressed_data[index + sub_index] == 0) {
                d_input_file_data[index / 8] = d_input_file_data[index / 8] << 1;
            } else {
                d_input_file_data[index / 8] = (d_input_file_data[index / 8] << 1) | 1;
            }
        }
    }

    // Calculate byte offset where overflow data should start in output
    const unsigned int offset_overflow = d_compressed_data_offset[overflow_position] / 8;

    // Phase 3: Bit packing for post-overflow data
    // Pack bits from d_temp_overflow into output buffer after the overflow offset
    for (index = pos * 8; index < d_compressed_data_offset[input_file_length]; index += blockDim.x * 8) {
        for (sub_index = 0; sub_index < 8; sub_index++) {
            if (d_temp_overflow[index + sub_index] == 0) {
                d_input_file_data[(index / 8) + offset_overflow] =
                        d_input_file_data[(index / 8) + offset_overflow] << 1;
            } else {
                d_input_file_data[(index / 8) + offset_overflow] =
                        (d_input_file_data[(index / 8) + offset_overflow] << 1) | 1;
            }
        }
    }
}

/**
 * @brief CUDA kernel for multi-run compression without integer overflow
 * @param d_input_file_data Device input data array
 * @param d_compressed_data_offset Device bit offset array
 * @param d_huffman_dictionary Device Huffman table
 * @param d_byte_compressed_data Device compressed bit buffer
 * @param d_lower_position Starting index for this chunk (inclusive)
 * @param const_memory_flag Constant memory usage flag
 * @param d_upper_position Ending index for this chunk (exclusive)
 *
 * This kernel processes a specific chunk of a large file in a multi-kernel approach.
 * Large files are divided into chunks to fit within GPU memory constraints.
 * Each kernel call processes one chunk sequentially.
 *
 * Special handling is needed for chunk boundaries to ensure bit sequences
 * that span chunk boundaries are properly encoded.
 */
__global__ void compress(unsigned char *d_input_file_data, const unsigned int *d_compressed_data_offset,
                         const struct huffman_dictionary *d_huffman_dictionary, unsigned char *d_byte_compressed_data,
                         const unsigned int d_lower_position, const unsigned int const_memory_flag,
                         const unsigned int d_upper_position) {
    // Load Huffman table into shared memory
    __shared__ struct huffman_dictionary table;
    memcpy(&table, d_huffman_dictionary, sizeof(struct huffman_dictionary));

    unsigned int index, bit_index;
    const unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Phase 1: Bit-level encoding for this chunk
    if (const_memory_flag == 0) {
        // Process bytes within the specified chunk range [d_lower_position, d_upper_position)
        for (index = pos + d_lower_position; index < d_upper_position; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                    d_input_file_data[index]][bit_index];
            }
        }

        // Handle chunk boundary condition
        // If this isn't the first chunk, need to encode the last byte of previous chunk
        // This ensures proper bit sequence continuity across chunk boundaries
        if (pos == 0 && d_lower_position != 0) {
            memcpy(
                &d_byte_compressed_data[d_compressed_data_offset[(d_lower_position)] - table.bit_sequence_length[
                                            d_input_file_data[d_lower_position - 1]]],
                &table.bit_sequence[d_input_file_data[d_lower_position - 1]],
                table.bit_sequence_length[d_input_file_data[d_lower_position - 1]]);
        }
    } else {
        // Hybrid memory approach for chunk processing
        for (index = pos + d_lower_position; index < d_upper_position; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                if (bit_index < 191) {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                        d_input_file_data[index]][
                        bit_index];
                } else {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = d_bit_sequence_const_memory[
                        d_input_file_data[index]][bit_index];
                }
            }
        }

        // Handle chunk boundary with constant memory
        if (pos == 0 && d_lower_position != 0) {
            memcpy(
                &d_byte_compressed_data[d_compressed_data_offset[(d_lower_position)] - table.bit_sequence_length[
                                            d_input_file_data[d_lower_position - 1]]],
                &d_bit_sequence_const_memory[d_input_file_data[d_lower_position - 1]],
                table.bit_sequence_length[d_input_file_data[d_lower_position - 1]]);
        }
    }

    // Synchronize before bit packing
    __syncthreads();

    // Phase 2: Bit packing for this chunk
    // Pack all bits generated by this chunk into bytes
    for (index = pos * 8; index < d_compressed_data_offset[d_upper_position]; index += blockDim.x * 8) {
        for (unsigned int sub_index = 0; sub_index < 8; sub_index++) {
            if (d_byte_compressed_data[index + sub_index] == 0) {
                d_input_file_data[(index / 8)] = d_input_file_data[(index / 8)] << 1;
            } else {
                d_input_file_data[(index / 8)] = (d_input_file_data[index / 8] << 1) | 1;
            }
        }
    }
}

/**
 * @brief CUDA kernel for multi-run compression with integer overflow handling
 * @param d_input_file_data Device input data array
 * @param d_compressed_data_offset Device bit offset array
 * @param d_huffman_dictionary Device Huffman table
 * @param d_byte_compressed_data Device buffer for pre-overflow bits
 * @param d_temp_overflow Device buffer for post-overflow bits
 * @param d_lower_position Start of chunk range
 * @param const_memory_flag Constant memory flag
 * @param d_upper_position End of chunk range
 * @param overflow_position Index where integer overflow occurs in this chunk
 *
 * This is the most complex compression scenario, combining:
 * - Multi-chunk processing for large files
 * - Integer overflow handling within chunks
 * - Chunk boundary management
 * - Dual buffer management for overflow data
 *
 * When a chunk itself experiences integer overflow, it must be split
 * into pre-overflow and post-overflow segments, each using different buffers.
 */
__global__ void compress(unsigned char *d_input_file_data, const unsigned int *d_compressed_data_offset,
                         const struct huffman_dictionary *d_huffman_dictionary, unsigned char *d_byte_compressed_data,
                         unsigned char *d_temp_overflow, const unsigned int d_lower_position,
                         const unsigned int const_memory_flag,
                         const unsigned int d_upper_position, const unsigned int overflow_position) {
    // Load Huffman table to shared memory
    __shared__ struct huffman_dictionary table;
    memcpy(&table, d_huffman_dictionary, sizeof(struct huffman_dictionary));

    unsigned int index, sub_index, bit_index;
    const unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Phase 1: Complex bit-level encoding with both chunk and overflow boundaries
    if (const_memory_flag == 0) {
        // Process chunk data before overflow point
        for (index = pos + d_lower_position; index < overflow_position; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                    d_input_file_data[index]][bit_index];
            }
        }

        // Process chunk data after overflow point
        for (index = overflow_position + pos; index < d_upper_position - 1; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index + 1]]; bit_index++) {
                d_temp_overflow[d_compressed_data_offset[index + 1] + bit_index] = table.bit_sequence[d_input_file_data[
                    index + 1]][bit_index];
            }
        }

        // Handle the overflow byte (thread 0 only)
        if (pos == 0) {
            memcpy(
                &d_temp_overflow[d_compressed_data_offset[(overflow_position + 1)] - table.bit_sequence_length[
                                     d_input_file_data[overflow_position]]],
                &table.bit_sequence[d_input_file_data[overflow_position]],
                table.bit_sequence_length[d_input_file_data[overflow_position]]);
        }

        // Handle chunk boundary (if not first chunk)
        if (pos == 0 && d_lower_position != 0) {
            memcpy(
                &d_byte_compressed_data[d_compressed_data_offset[(d_lower_position)] - table.bit_sequence_length[
                                            d_input_file_data[d_lower_position - 1]]],
                &table.bit_sequence[d_input_file_data[d_lower_position - 1]],
                table.bit_sequence_length[d_input_file_data[d_lower_position - 1]]);
        }
    } else {
        // Hybrid memory approach with complex boundary handling
        // Process entire chunk range, but handle overflow internally
        for (index = pos + d_lower_position; index < d_upper_position; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index]]; bit_index++) {
                if (bit_index < 191) {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                        d_input_file_data[index]][
                        bit_index];
                } else {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = d_bit_sequence_const_memory[
                        d_input_file_data[index]][bit_index];
                }
            }
        }

        // Process post-overflow data separately
        for (index = overflow_position + pos; index < d_upper_position - 1; index += blockDim.x) {
            for (bit_index = 0; bit_index < table.bit_sequence_length[d_input_file_data[index + 1]]; bit_index++) {
                if (bit_index < 191) {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = table.bit_sequence[
                        d_input_file_data[index]][
                        bit_index];
                } else {
                    d_byte_compressed_data[d_compressed_data_offset[index] + bit_index] = d_bit_sequence_const_memory[
                        d_input_file_data[index]][bit_index];
                }
            }
        }

        // Handle overflow byte with constant memory
        if (pos == 0) {
            memcpy(
                &d_temp_overflow[d_compressed_data_offset[(overflow_position + 1)] - table.bit_sequence_length[
                                     d_input_file_data[overflow_position]]],
                &d_bit_sequence_const_memory[d_input_file_data[overflow_position]],
                table.bit_sequence_length[d_input_file_data[overflow_position]]);
        }

        // Handle chunk boundary with constant memory
        if (pos == 0 && d_lower_position != 0) {
            memcpy(
                &d_byte_compressed_data[d_compressed_data_offset[(d_lower_position)] - table.bit_sequence_length[
                                            d_input_file_data[d_lower_position - 1]]],
                &d_bit_sequence_const_memory[d_input_file_data[d_lower_position - 1]],
                table.bit_sequence_length[d_input_file_data[d_lower_position - 1]]);
        }
    }

    // Synchronize before bit packing phase
    __syncthreads();

    // Phase 2: Bit packing for pre-overflow segment
    for (index = pos * 8; index < d_compressed_data_offset[overflow_position]; index += blockDim.x * 8) {
        for (sub_index = 0; sub_index < 8; sub_index++) {
            if (d_byte_compressed_data[index + sub_index] == 0) {
                d_input_file_data[(index / 8)] = d_input_file_data[(index / 8)] << 1;
            } else {
                d_input_file_data[(index / 8)] = (d_input_file_data[index / 8] << 1) | 1;
            }
        }
    }

    // Calculate overflow offset for this chunk
    const unsigned int offset_overflow = d_compressed_data_offset[overflow_position] / 8;

    // Phase 3: Bit packing for post-overflow segment
    for (index = pos * 8; index < d_compressed_data_offset[d_upper_position]; index += blockDim.x * 8) {
        for (sub_index = 0; sub_index < 8; sub_index++) {
            if (d_temp_overflow[index + sub_index] == 0) {
                d_input_file_data[(index / 8) + offset_overflow] =
                        d_input_file_data[(index / 8) + offset_overflow] << 1;
            } else {
                d_input_file_data[(index / 8) + offset_overflow] =
                        (d_input_file_data[(index / 8) + offset_overflow] << 1) | 1;
            }
        }
    }
}
